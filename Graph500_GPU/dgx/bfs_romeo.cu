#include "hip/hip_runtime.h"
#include "../compat.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C"{
#include "../graph500.h"
}

#include "../xalloc.h"
#include "../generator/graph_generator.h"

// Set by Makefile 
#define NWARPS 8

//#define CHUNK_ELEM 4096
//#define CHUNK_SIZE (sizeof(int32_t)*CHUNK_ELEM) // Size of a chunk in byte

#define BITMAP_TYPE uint32_t
#define BITMAP_WORD 32

/* Global variables */
static int64_t maxvtx; // total number of vertices 
static int64_t nv; // number of vertices 
static int64_t maxedg;
static int32_t nwords; 

/* Host pointers */
static int32_t * h_CSR_R;
static int32_t * h_CSR_C;
static int32_t * h_predecessors;
static int32_t h_n_in_queue;
static int32_t h_n_out_queue;

/* Device pointers */
static int32_t * d_CSR_R;
static int32_t * d_CSR_C;
static BITMAP_TYPE * d_in_queue;
static BITMAP_TYPE * d_out_queue;
static int32_t * d_predecessors; 
static int32_t * d_n_in_queue; 
static int32_t * d_n_out_queue;
static BITMAP_TYPE * d_visited_tex;
__constant__ int32_t d_nwords;

static hipEvent_t start, stop;

static void HandleError(hipError_t err, 
			const char *file,
			int line)
{
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d \n",hipGetErrorString(err),file,line);
		exit(EXIT_FAILURE); 
	}
}

#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))

/* "Private functions" */
/* Compute the total number of vertices in the generated graph */
static void find_nv(const struct packed_edge * restrict IJ, const int64_t nedge)
{
	maxvtx = -1;
	// Here use the 40 cores to compute   
	#pragma omp parallel 
	{
		int64_t k;
		#pragma omp for reduction(max:maxvtx)
		for(k = 0 ; k < nedge ; ++k)
		{
			if(get_v0_from_edge(&IJ[k]) > maxvtx)
				maxvtx = get_v0_from_edge(&IJ[k]);
			if(get_v1_from_edge(&IJ[k]) > maxvtx)
				maxvtx = get_v1_from_edge(&IJ[k]);
		}
	} 
	nv = maxvtx+1;
}

void 
omp_prefix_sum(int32_t * x, int N)
{
	int32_t * suma; 
	#pragma omp parallel 
	{
		const int ithread = omp_get_thread_num();
		const int nthreads = omp_get_num_threads();
		#pragma omp single
		{	
			suma = (int32_t*)malloc(sizeof(int32_t)*nthreads+1);
			suma[0] = 0;
		}
		int32_t sum = 0;
		#pragma omp for schedule(static)
		for(unsigned int i = 0 ; i < N ; ++i)
		{
			sum += x[i];
			x[i] = sum; 
		}
		suma[ithread+1] = sum; 
		#pragma omp barrier
		float offset = 0;
		for(unsigned int i = 0 ; i < (ithread+1) ; ++i)
			offset += suma[i];
		#pragma omp for schedule(static)
		for(unsigned int i = 0 ; i < N ; ++i)
			x[i] += offset;
	}
	
	for(unsigned int i = N ; i > 0 ; --i)
		x[i] = x[i-1];
	x[0] = 0;

	free(suma);

}

static void 
edgelist_to_CSR(const struct packed_edge * restrict IJ, const int64_t nedge)
{
	//int32_t *h_chunk_v0, *h_chunk_v1; 
	//int32_t *d_chunk_v0, *d_chunk_v1;
 
	//int nchunk = (2*nedge*sizeof(int32_t))/CHUNK_SIZE;

	//printf("MAXVTX(%" PRId64 ")\n",maxvtx);
	//printf("NV(%" PRId64 ")\n",nv);

	hipSetDevice(0);
	/* Init CSR arrays on GPU */
	HANDLE_ERROR(hipMalloc((void**)&d_CSR_R,sizeof(int32_t)*(nv+1)));
	h_CSR_R = (int32_t*)malloc(sizeof(int32_t)*(nv+1));
	assert(h_CSR_R); 
	memset(h_CSR_R,0,sizeof(int32_t)*(nv+1));

	/* Step one, count the CSR_R and CSR_C size */
	maxedg = 0;

	#pragma omp parallel for reduction(+:maxedg)
	for(unsigned int i = 0 ; i < nedge ; ++i)
	{
		// No self loop 
		if(get_v0_from_edge(&IJ[i]) != get_v1_from_edge(&IJ[i]))
		{
			__sync_fetch_and_add(&h_CSR_R[get_v0_from_edge(&IJ[i])],1);
			__sync_fetch_and_add(&h_CSR_R[get_v1_from_edge(&IJ[i])],1);
			maxedg+=2;
		}
	}
	
	//printf("MAXEDG(%" PRId64 ")\n",maxedg);

	int32_t tot = 0;
	for(unsigned int i = 0 ; i < nv+1 ; ++i)
		tot += h_CSR_R[i];
	printf("tot(%d)\n",tot);
	
	// Malloc CRC array 
	h_CSR_C = (int32_t*)malloc(sizeof(int32_t)*maxedg);
	assert(h_CSR_C);
	HANDLE_ERROR(hipMalloc((void**)&d_CSR_C,sizeof(int32_t)*maxedg));

	//omp_prefix_sum(h_CSR_R,nv);
	int32_t tmp = h_CSR_R[0];
	for(unsigned int i = 1 ; i < nv+1 ; ++i)
	{
		int32_t tmp2 = h_CSR_R[i];
		h_CSR_R[i] = tmp;
		tmp += tmp2;
	}
	h_CSR_R[0] = 0;

	printf("last(%d)\n",h_CSR_R[nv]);
	assert(h_CSR_R[nv] == maxedg);
	//printf("\nCSR_R list");
	//for(unsigned int i = 0 ; i < nv-1 ; ++i)
	//	printf(" %d(%d)",h_CSR_R[i],h_CSR_R[i+1] - h_CSR_R[i]);
	//printf("\n");

	int32_t * CSR_R_counter = (int32_t*)malloc(sizeof(int32_t)*nv); 
	assert(CSR_R_counter);
	memset(CSR_R_counter,0,sizeof(int32_t)*nv);

	//printf("CSR_C generiation\n");

	/* Step two generate CSC array */
	#pragma omp parallel for 
	for(unsigned int i = 0 ; i < nedge ; ++i)

	{
		int32_t v0 = (int32_t)get_v0_from_edge(&IJ[i]);
		int32_t v1 = (int32_t)get_v1_from_edge(&IJ[i]);
		if(v0 != v1)
		{
			int counter_v0 = __sync_fetch_and_add(&(CSR_R_counter[v0]),1);
			int counter_v1 = __sync_fetch_and_add(&(CSR_R_counter[v1]),1);
			//printf("Edge(%d,%d) added in %d(%d) and %d(%d)\n",v0,v1,v0,counter_v0,v1,counter_v1);
			h_CSR_C[h_CSR_R[v0]+counter_v0] = v1;	
			h_CSR_C[h_CSR_R[v1]+counter_v1] = v0;
		 }
	}
	free(CSR_R_counter);
	//printf("\nMalloc\n");
	// Copy CSR and CSC on GPU 
	HANDLE_ERROR(hipMemcpy(d_CSR_R,h_CSR_R,sizeof(int32_t)*(nv+1),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_CSR_C,h_CSR_C,sizeof(int32_t)*maxedg,hipMemcpyHostToDevice));

	// Prepare in and ou queues as bitmap 
	nwords = (nv + (BITMAP_WORD / 2)) / BITMAP_WORD; 
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_nwords),&nwords,sizeof(int32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_in_queue,sizeof(BITMAP_TYPE)*words));
	HANDLE_ERROR(hipMalloc((void**)&d_out_queue,sizeof(BITMAP_TYPE)*words));
	HANDLE_ERROR(hipMalloc((void**)&d_n_in_queue,sizeof(int32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_n_out_queue,sizeof(int32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_visited_tex,sizeof(BITMAP_TYPE)*nwords));	

	HANDLE_ERROR(hipMalloc((void**)&d_predecessors,sizeof(int32_t)*nv));
	h_predecessors = (int32_t*)malloc(sizeof(int32_t)*nv);
	assert(h_predecessors);

	hipEventCreate(&start);
	hipEventCreate(&stop); 

	printf("End make_CSR\n");

}

__inline__ __device__ int warpScanSumDown(int val)
{
	int lane_id = threadIdx.x & 31; 
	for(int offset = 1 ; offset < 32 ; offset <<= 1)
	{
		int y = __shfl_down(val,offset); 
		if(lane_id <= 31 - offset)
			val += y;
	}
	return val; 
}

__inline__ __device__ int warpScanSum(int val)
{
	int lane_id = threadIdx.x & 31; 
	for(int offset = 1 ; offset < 32 ; offset <<= 1)
	{
		int y = __shfl_up(val,offset); 	
		if(lane_id >= offset)
			val += y;
	}
	return val; 
}


__inline__ __device__ int warpReduceSum(int val)
{
	for(int offset = warpSize/2 ; offset > 0 ; offset /= 2)
		val += __shfl_down(val,offset);
	return val;
} 

__global__ void explore_frontier_CSR( BITMAP_TYPE * out_queue,  int32_t * visited_label, BITMAP_TYPE * visited_tex,  uint32_t * n_out_queue, unsigned int * R, unsigned int * C)
{
	int lane_id = threadIdx.x & 31;
	int warp_id = threadIdx.x >> 5;
	
	int ligne = threadIdx.x+blockIdx.x*blockDim.x;
	int32_t value_visited = visited_label[ligne];			//GLOBAL
	int actif = 0;
	if(value_visited == -1)
		actif = 1;
		
	if(!__any(actif))
		return;
		
	unsigned int word = ligne/BITMAP_WORD;
	unsigned int range[3] = {0,0,0};
	
	if(value_visited == -1)
	{
		range[0] = R[ligne];
		range[1] = R[ligne+1];
		range[2] = range[1] - range[0];
	}
	
	// On va explorer chaque ligne successivement 
	volatile __shared__ int comm[NWARPS][3];
	volatile __shared__ int shared_ligne[NWARPS];
	volatile __shared__ int sum[NWARPS];
	volatile __shared__ int fin[NWARPS];
	
	if(lane_id == 0)
		sum[warp_id] = 0;
	
	while( __any(range[2]) )
	{
		int voisin = -1;
	
		if(range[2])
			comm[warp_id][0] = lane_id;
	
		if(comm[warp_id][0] == lane_id)
		{
			comm[warp_id][1] = range[0];
			comm[warp_id][2] = range[1];
			range[2] = 0;
			shared_ligne[warp_id] = ligne;
		}
		
		int r_gather = comm[warp_id][1] + lane_id;
		int r_gather_end = comm[warp_id][2];
		
		if(lane_id==0)
			fin[warp_id] = 0;
		
		while(r_gather < r_gather_end && !fin[warp_id])
		{
			voisin = C[r_gather];
	
			// Vérifier voisin dans in_queue
			unsigned int position = voisin / BITMAP_WORD;
			BITMAP_TYPE mask = tex1Dfetch(tex_in_queue,position);
			BITMAP_TYPE mask_bit = 1 << (voisin % BITMAP_WORD);
			if(mask & mask_bit)
			{
				// Ajout direct du voisin dans visited et passer à la suite 
				//visited_label[shared_ligne[warp_id]] = voisin+d_offset;
				//int old = atomicCAS(&visited_label[shared_ligne[warp_id]],-1,voisin+d_offset);
				//if(old == -1)

				visited_label[shared_ligne[warp_id]] =  voisin+d_offset;
				if(visited_label[shared_ligne[warp_id]] == voisin+d_offset)
				{
					visited_tex[word] |= 1 << shared_ligne[warp_id]%BITMAP_WORD;
					out_queue[word] |= 1 << shared_ligne[warp_id]%BITMAP_WORD;
					++sum[warp_id];
					fin[warp_id] = 1;
				}
			}
			r_gather+=32;
		}
	}
	
	if(lane_id == 0 && sum[warp_id])
		atomicAdd(n_out_queue,sum[warp_id]);
}

//__launch_bounds__(NWARPS*32, MIN_BLOCKS_PER_SMX)
__global__ void explore_frontier_CSC( restrict BITMAP_TYPE * in_queue, restrict BITMAP_TYPE * out_queue,  int32_t * visited_label, BITMAP_TYPE * visited_tex , uint32_t * n_out_queue, unsigned int * R, unsigned int * C)
{
	int lane_id = threadIdx.x & 31;
	int warp_id = threadIdx.x >> 5; 

	int word = blockIdx.x*NWARPS+warp_id;
	int val_in_queue = in_queue[word];								// GLOBAL
	if(val_in_queue == 0)
		return;

	int id_sommet = -1;
	unsigned int range[3] = {0,0,0};
	
	if(val_in_queue & 1 << lane_id)
	{
		id_sommet = word*32+lane_id;
		range[0] = C[id_sommet];									//GLOBAL
		range[1] = C[id_sommet+1];								//GLOBAL
		range[2] = range[1] - range[0];
	}

	volatile __shared__ int comm[NWARPS][3];
	volatile __shared__ int shared_sommet[NWARPS];
	uint32_t sum;

	while( __any(range[2]) )
	{

		int voisin = -1;

		if(range[2])
			comm[warp_id][0] = lane_id;							// SHARED

		if(comm[warp_id][0] == lane_id)
		{
			comm[warp_id][1] = range[0];							// SHARED
			comm[warp_id][2] = range[1];							// SHARED
			range[2] = 0;
			shared_sommet[warp_id] = id_sommet;					// SHARED
		}

		int r_gather = comm[warp_id][1] + lane_id;
		int r_gather_end = comm[warp_id][2];
		while(r_gather < r_gather_end)
		{
			sum = 0;
			voisin = R[r_gather];								// GLOBAL

			unsigned int position = voisin / BITMAP_WORD;
			BITMAP_TYPE mask = tex1Dfetch(tex_visited,position);
			BITMAP_TYPE mask_bit = 1 << (voisin % BITMAP_WORD);
			if(!(mask & mask_bit))
			{
				visited_tex[position] |= mask_bit;
				//int32_t value = atomicCAS(&visited_label[voisin],-1,shared_sommet[warp_id]+d_offset);
				if(visited_label[voisin] == -1)
					visited_label[voisin] = shared_sommet[warp_id]+d_offset;

				if(visited_label[voisin] == shared_sommet[warp_id]+d_offset)
				{
					unsigned int val_out_queue = 1 << voisin%32;  
					atomicOr(&out_queue[voisin/32],val_out_queue);
					sum = 1;
				}
			}

			// TODO faire à la fin 
			if(__any(sum))
			{
				sum = warpReduceSum(sum);
				if(lane_id == 0)
					atomicAdd(n_out_queue,sum);
			}

			r_gather+=32;
		}

	}
}


__global__ void setup_GPU(int32_t * predecessors, int64_t srcvtx, BITMAP_TYPE * in_queue, BITMAP_TYPE * visited_tex)
{
	predecessors[srcvtx] = (int32_t)srcvtx;
	in_queue[srcvtx/BITMAP_WORD] = srcvtx%BITMAP_WORD;
	in_queue[srcvtx/BITMAP_WORD] = srcvtx%BITMAP_WORD;
}

/* Create the graph structure on the GPUs */
extern "C"
int create_graph_from_edgelist(struct packed_edge * IJ, int64_t nedge)
{
	//printf("create_graph_from_edgelist nedge(%" PRId64 ")\n",nedge);
	#pragma omp parallel 
	#pragma omp single 
	printf("%d threads\n", omp_get_num_threads());
	/* Each thread handle a GPU */		
	find_nv(IJ,nedge);

	/* Compute CSR representation */
	edgelist_to_CSR(IJ,nedge);

	return 0; 
}

extern "C"
int make_bfs_tree( int64_t *bfs_tree_out, int64_t *max_vtx_out, int64_t srcvtx)
{
	printf("\n");
	// TODO check this nv != maxvtx
	*max_vtx_out = maxvtx;
	h_n_in_queue = 1;
	HANDLE_ERROR(hipMemcpy(d_n_in_queue,&h_n_in_queue,sizeof(int32_t),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(d_in_queue,0,sizeof(BITMAP_TYPE)*nwords));
	HANDLE_ERROR(hipMemset(d_visited_tex,0,sizeof(BITMAP_TYPE*nwords)));
	HANDLE_ERROR(hipMemset(d_predecessors,-1,sizeof(int32_t)*nv));
	setup_GPU<<<1,1>>>(d_predecessors,srcvtx,d_in_queue,d_visited_tex);
	//setup_GPU<<<(nv + (NWARPS/2))/NWARPS,NWARPS>>>(d_predecessors,srcvtx);
	
	int32_t iteration = 0;

	while(1)
	{
		if(iteration++ > 1 << 20)
		{
			fprintf(stderr,"Too many iterations(%d)\n",iteration);
			return -1;
		}

		dim3 dimGrid(h_n_in_queue/(NWARPS*32)+1,0,0); 		
		dim3 dimBlock(32*NWARPS ,0,0);
	
		printf("iteration(%2d) n_in_queue(%10d) nblocks(%4d) nthreads(%d) ",
			iteration,
			h_n_in_queue,
			dimGrid.x,
			dimBlock.x);
		fflush(stdout);
	
		HANDLE_ERROR(hipMemset(d_n_out_queue,0,sizeof(int32_t)));
		hipEventRecord(start);
		

		if(curlevel < 3)		
		{
			HANDLE_ERROR(hipMemset(d_out_queue,0,sizeof(BITMAP_TYPE)*nwords));
			HANDLE_ERROR(hipBindTexture(0, tex_visited, d_visited_tex,sizeof(BITMAP_TYPE)*nwords));

			explore_frontier_CSC<<< nwords/NWARPS , 32*NWARPS >>>( d_in_queue, d_out_queue, d_visited_label,d_visited_tex, d_n_out_queue, d_CSR_R, d_CSR_C);
			HANDLE_ERROR(hipMemcpy(&h_n_out_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipUnbindTexture(tex_visited));
		}else{
			HANDLE_ERROR(hipMemset(d_n_out_queue,0,sizeof(uint32_t)));
			HANDLE_ERROR(hipMemset(d_out_queue,0,sizeof(BITMAP_TYPE)*nwords));
			HANDLE_ERROR(hipBindTexture(0, tex_in_queue, d_in_queue,sizeof(BITMAP_TYPE)*nwords));

			explore_frontier_CSR<<< nwords/NWARPS , 32*NWARPS >>>(d_out_queue, d_visited_label,d_visited_tex, d_n_out_queue, d_CSR_R, d_CSR_C);
			HANDLE_ERROR(hipMemcpy(&h_n_out_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipUnbindTexture(tex_in_queue));
		}



		hipEventRecord(stop);
		HANDLE_ERROR(hipMemcpy(&h_n_out_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToHost));
		hipEventSynchronize(stop);
		float milliseconds = 0; 
		hipEventElapsedTime(&milliseconds,start,stop);
		printf("out_queue(%10d) time(%.4f)s \n",h_n_out_queue,milliseconds/1000);	
		if(h_n_out_queue == 0)
		{
			printf("BFS ended\n");
			break;
		}	
		/* Switch queues */
		HANDLE_ERROR(hipMemcpy(d_in_queue,d_out_queue,sizeof(BITMAP_TYPE)*nwords,hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(d_n_in_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToDevice));
		h_n_in_queue = h_n_out_queue;
	}

	HANDLE_ERROR(hipMemcpy(h_predecessors,d_predecessors,sizeof(int32_t)*nv,hipMemcpyDeviceToHost));
	#pragma omp parallel for
	for(unsigned int i = 0 ; i < nv ; ++i)
	{
		bfs_tree_out[i] = (int64_t)h_predecessors[i];		
		assert(bfs_tree_out[i] < nv);
		assert(bfs_tree_out[i] > -2);
	}
	return 0;
}

extern "C"
void destroy_graph()
{
	free(h_CSR_R);
	free(h_CSR_C);
	hipFree(d_CSR_R);
	hipFree(d_CSR_C);
	hipFree(d_in_queue);
	hipFree(d_out_queue);
}
