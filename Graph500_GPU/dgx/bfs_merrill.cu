#include "hip/hip_runtime.h"
#include "../compat.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C"{
#include "../graph500.h"
}

#include "../xalloc.h"
#include "../generator/graph_generator.h"

// Set by Makefile 
#define NWARPS 8

//#define CHUNK_ELEM 4096
//#define CHUNK_SIZE (sizeof(int32_t)*CHUNK_ELEM) // Size of a chunk in byte

#define BITMAP_TYPE uint32_t
#define BITMAP_WORD 32

/* Global variables */
static int64_t maxvtx; // total number of vertices 
static int64_t nv; // number of vertices 
static int64_t maxedg;
static int32_t nwords; 

/* Host pointers */
static int32_t * h_CSR_R;
static int32_t * h_CSR_C;
static int32_t * h_predecessors;
static int32_t h_n_in_queue;
static int32_t h_n_out_queue;

/* Device pointers */
static int32_t * d_CSR_R;
static int32_t * d_CSR_C;
static int32_t * d_in_queue;
static int32_t * d_out_queue;
static int32_t * d_predecessors; 
static int32_t * d_n_in_queue; 
static int32_t * d_n_out_queue;
__constant__ int32_t d_nwords;

static hipEvent_t start, stop;

static void HandleError(hipError_t err, 
			const char *file,
			int line)
{
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d \n",hipGetErrorString(err),file,line);
		exit(EXIT_FAILURE); 
	}
}

#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))

/* "Private functions" */
/* Compute the total number of vertices in the generated graph */
static void find_nv(const struct packed_edge * restrict IJ, const int64_t nedge)
{
	maxvtx = -1;
	// Here use the 40 cores to compute   
	#pragma omp parallel 
	{
		int64_t k;
		#pragma omp for reduction(max:maxvtx)
		for(k = 0 ; k < nedge ; ++k)
		{
			if(get_v0_from_edge(&IJ[k]) > maxvtx)
				maxvtx = get_v0_from_edge(&IJ[k]);
			if(get_v1_from_edge(&IJ[k]) > maxvtx)
				maxvtx = get_v1_from_edge(&IJ[k]);
		}
	} 
	nv = maxvtx+1;
}

void 
omp_prefix_sum(int32_t * x, int N)
{
	int32_t * suma; 
	#pragma omp parallel 
	{
		const int ithread = omp_get_thread_num();
		const int nthreads = omp_get_num_threads();
		#pragma omp single
		{	
			suma = (int32_t*)malloc(sizeof(int32_t)*nthreads+1);
			suma[0] = 0;
		}
		int32_t sum = 0;
		#pragma omp for schedule(static)
		for(unsigned int i = 0 ; i < N ; ++i)
		{
			sum += x[i];
			x[i] = sum; 
		}
		suma[ithread+1] = sum; 
		#pragma omp barrier
		float offset = 0;
		for(unsigned int i = 0 ; i < (ithread+1) ; ++i)
			offset += suma[i];
		#pragma omp for schedule(static)
		for(unsigned int i = 0 ; i < N ; ++i)
			x[i] += offset;
	}
	
	for(unsigned int i = N ; i > 0 ; --i)
		x[i] = x[i-1];
	x[0] = 0;

	free(suma);

}

static void 
edgelist_to_CSR(const struct packed_edge * restrict IJ, const int64_t nedge)
{
	//int32_t *h_chunk_v0, *h_chunk_v1; 
	//int32_t *d_chunk_v0, *d_chunk_v1;
 
	//int nchunk = (2*nedge*sizeof(int32_t))/CHUNK_SIZE;

	//printf("MAXVTX(%" PRId64 ")\n",maxvtx);
	//printf("NV(%" PRId64 ")\n",nv);

	hipSetDevice(0);
	/* Init CSR arrays on GPU */
	HANDLE_ERROR(hipMalloc((void**)&d_CSR_R,sizeof(int32_t)*(nv+1)));
	h_CSR_R = (int32_t*)malloc(sizeof(int32_t)*(nv+1));
	assert(h_CSR_R); 
	memset(h_CSR_R,0,sizeof(int32_t)*(nv+1));

	/* Step one, count the CSR_R and CSR_C size */
	maxedg = 0;

	#pragma omp parallel for reduction(+:maxedg)
	for(unsigned int i = 0 ; i < nedge ; ++i)
	{
		// No self loop 
		if(get_v0_from_edge(&IJ[i]) != get_v1_from_edge(&IJ[i]))
		{
			__sync_fetch_and_add(&h_CSR_R[get_v0_from_edge(&IJ[i])],1);
			__sync_fetch_and_add(&h_CSR_R[get_v1_from_edge(&IJ[i])],1);
			maxedg+=2;
		}
	}
	
	//printf("MAXEDG(%" PRId64 ")\n",maxedg);

	int32_t tot = 0;
	for(unsigned int i = 0 ; i < nv+1 ; ++i)
		tot += h_CSR_R[i];
	printf("tot(%d)\n",tot);
	
	// Malloc CRC array 
	h_CSR_C = (int32_t*)malloc(sizeof(int32_t)*maxedg);
	assert(h_CSR_C);
	HANDLE_ERROR(hipMalloc((void**)&d_CSR_C,sizeof(int32_t)*maxedg));

	//omp_prefix_sum(h_CSR_R,nv);
	int32_t tmp = h_CSR_R[0];
	for(unsigned int i = 1 ; i < nv+1 ; ++i)
	{
		int32_t tmp2 = h_CSR_R[i];
		h_CSR_R[i] = tmp;
		tmp += tmp2;
	}
	h_CSR_R[0] = 0;

	printf("last(%d)\n",h_CSR_R[nv]);
	assert(h_CSR_R[nv] == maxedg);
	//printf("\nCSR_R list");
	//for(unsigned int i = 0 ; i < nv-1 ; ++i)
	//	printf(" %d(%d)",h_CSR_R[i],h_CSR_R[i+1] - h_CSR_R[i]);
	//printf("\n");

	int32_t * CSR_R_counter = (int32_t*)malloc(sizeof(int32_t)*nv); 
	assert(CSR_R_counter);
	memset(CSR_R_counter,0,sizeof(int32_t)*nv);

	//printf("CSR_C generiation\n");

	/* Step two generate CSC array */
	#pragma omp parallel for 
	for(unsigned int i = 0 ; i < nedge ; ++i)

	{
		int32_t v0 = (int32_t)get_v0_from_edge(&IJ[i]);
		int32_t v1 = (int32_t)get_v1_from_edge(&IJ[i]);
		if(v0 != v1)
		{
			int counter_v0 = __sync_fetch_and_add(&(CSR_R_counter[v0]),1);
			int counter_v1 = __sync_fetch_and_add(&(CSR_R_counter[v1]),1);
			//printf("Edge(%d,%d) added in %d(%d) and %d(%d)\n",v0,v1,v0,counter_v0,v1,counter_v1);
			h_CSR_C[h_CSR_R[v0]+counter_v0] = v1;	
			h_CSR_C[h_CSR_R[v1]+counter_v1] = v0;
		 }
	}
	free(CSR_R_counter);
	//printf("\nMalloc\n");
	// Copy CSR and CSC on GPU 
	HANDLE_ERROR(hipMemcpy(d_CSR_R,h_CSR_R,sizeof(int32_t)*(nv+1),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_CSR_C,h_CSR_C,sizeof(int32_t)*maxedg,hipMemcpyHostToDevice));

	// Prepare in and ou queues as bitmap 
	//nwords = (nv + (BITMAP_WORD / 2)) / BITMAP_WORD; 
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_nwords),&nwords,sizeof(int32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_in_queue,sizeof(int32_t)*maxedg));
	HANDLE_ERROR(hipMalloc((void**)&d_out_queue,sizeof(int32_t)*maxedg));
	HANDLE_ERROR(hipMalloc((void**)&d_n_in_queue,sizeof(int32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_n_out_queue,sizeof(int32_t)));	

	HANDLE_ERROR(hipMalloc((void**)&d_predecessors,sizeof(int32_t)*nv));
	h_predecessors = (int32_t*)malloc(sizeof(int32_t)*nv);
	assert(h_predecessors);

	hipEventCreate(&start);
	hipEventCreate(&stop); 

	printf("End make_CSR\n");

}

__inline__ __device__ int warpScanSumDown(int val)
{
	int lane_id = threadIdx.x & 31; 
	for(int offset = 1 ; offset < 32 ; offset <<= 1)
	{
		int y = __shfl_down(val,offset); 
		if(lane_id <= 31 - offset)
			val += y;
	}
	return val; 
}

__inline__ __device__ int warpScanSum(int val)
{
	int lane_id = threadIdx.x & 31; 
	for(int offset = 1 ; offset < 32 ; offset <<= 1)
	{
		int y = __shfl_up(val,offset); 	
		if(lane_id >= offset)
			val += y;
	}
	return val; 
}


__inline__ __device__ int warpReduceSum(int val)
{
	for(int offset = warpSize/2 ; offset > 0 ; offset /= 2)
		val += __shfl_down(val,offset);
	return val;
} 

__global__ void explore_CSC(	int32_t * CSC_C, int32_t * CSC_R,
				BITMAP_TYPE * in_queue, int32_t * n_in_queue, 
				BITMAP_TYPE * out_queue, int32_t * n_out_queue, 
				int32_t * predecessors)
{
	int lane_id = threadIdx.x & 31; 
	int warp_id = threadIdx.x >> 5; 

	int vtx = threadIdx.x+blockIdx.x*blockDim.x;
	int32_t value_vtx = predecessors[vtx]; 
	int active = 0; 
	if(value_vtx == -1)
		active = 1;

	// No more active thread in this warp 
	if(!__any(active))
		return;

	int32_t range[3] = {0,0,0};

	if(active)
	{
		range[0] = CSC_C[vtx];
		range[1] = CSC_C[vtx+1]; 
		range[2] = range[1] - range[0];
	}

	volatile __shared__ int comm[NWARPS][3]; 
	volatile __shared__ int shared_vtx[NWARPS]; 
	volatile __shared__ int sum[NWARPS]; 
	volatile __shared__ int end[NWARPS]; 

	if(lane_id == 0)
		sum[warp_id] = 0;

	while(__any(range[2]))
	{
		int neighbor = -1; 

		if(range[2])
			comm[warp_id][0] = lane_id;
		if(comm[warp_id][0] == lane_id)
		{
			comm[warp_id][1] = range[0]; 
			comm[warp_id][2] = range[1]; 
			range[2] = 0; 
			shared_vtx[warp_id] = vtx; 
		}
		
		int r_gather = comm[warp_id][1] + lane_id;
		int r_gather_end = comm[warp_id][2]; 

		if(lane_id == 0)
			end[warp_id] = 0;

		while((r_gather < r_gather_end) && !end[warp_id])
		{
			neighbor = CSC_R[r_gather];
		
			BITMAP_TYPE value_in_queue = in_queue[neighbor/BITMAP_WORD];
			if(value_in_queue & (1<<(neighbor%BITMAP_WORD)))
			{
				// Can be changed because just this warp access to this vertex
				int prev = atomicCAS(&(predecessors[shared_vtx[warp_id]]),-1,neighbor);
				if(prev == -1)
				{
					int value_out_queue = 1 << (shared_vtx[warp_id]%BITMAP_WORD);
					atomicOr(&(out_queue[shared_vtx[warp_id]/BITMAP_WORD]),value_out_queue);
					++sum[warp_id];
					end[warp_id] = 1;
				}
			}			
			r_gather += 32; 
		}
	}
	if(lane_id == 0 && sum[warp_id])
		atomicAdd(n_out_queue,sum[warp_id]);
}

__device__ int warp_culling(int32_t neighbor)
{	
	int warp_id = threadIdx.x >> 5; 
	volatile __shared__ int scratch[NWARPS][128]; 
	int hash = neighbor & 127; 
	scratch[warp_id][hash] = neighbor; 
	int retrieved = scratch[warp_id][hash];
	if(retrieved == neighbor)
	{
		scratch[warp_id][hash]  = threadIdx.x; 
		if(scratch[warp_id][hash] != threadIdx.x)
			return true;
	}
	return false; 
}

__device__ int history_culling(int32_t neighbor)
{
	volatile __shared__ int cache[2048]; 
	// Initialize cache 
	int chunk = 2048 / blockDim.x;
	for(int i = threadIdx.x*chunk ; i < threadIdx.x*chunk + chunk ; i++)
		cache[i] = -1;
	__syncthreads(); 
	int hash = neighbor & 2047;
	int retrieved = cache[hash]; 
	if(retrieved == neighbor)
		return true; 
	cache[hash] = neighbor; 
	return false; 
}

__global__ void expand_contract_merrill(	int32_t * CSR_R, int32_t * CSR_C,
						int32_t * in_queue, int32_t * n_in_queue,
						int32_t * out_queue, int32_t * n_out_queue, 
						int32_t * predecessors)
{
	// Global thread id 
	int thx = threadIdx.x + blockIdx.x * blockDim.x;
	// Thread id in the warp 
	int lane_id = threadIdx.x & 31; 
	// Warp id in the block 
	int warp_id = threadIdx.x >> 5; 
	// vertex loaded in the in_queue
	int32_t vtx = -1;


	// Load one vtx from the in_queue 
	if(thx < *n_in_queue)
	{
		// GLOBAL MEMORY access
		vtx = in_queue[thx]; 
		// Check for duplicates 
		if(warp_culling(vtx))
			vtx = -1;
		//if(history_culling(vtx))
		//	vtx = -1;
	}
	
	//printf("thx(%d) load(%d)\n",thx,vtx);
	
	// Here every thread have a unique vtx or -1
	// Now load the adjacency list for each vtx
	int32_t range[3] = {0,0,0}; 
	if(vtx != -1)	
	{
		// 2 GLOBAL MEMORY accesses
		range[0] = CSR_R[vtx]; 
		range[1] = CSR_R[vtx+1]; 
		range[2] = range[1] - range[0];
	}

	volatile __shared__ int comm[NWARPS][3]; 
	volatile __shared__ int32_t shared_vertex[NWARPS]; 
	volatile __shared__ int offset_warp[NWARPS]; 

	while(__any(range[2]))
	{
		if(range[2])
			comm[warp_id][0] = lane_id; 

		if(comm[warp_id][0] == lane_id)
		{
			comm[warp_id][1] = range[0]; 
			comm[warp_id][2] = range[1]; 
			range[2] = 0;
			shared_vertex[warp_id] = vtx; 
		}

		int r_gather = comm[warp_id][1] + lane_id; 
		int r_gather_end = comm[warp_id][2]; 
		while(r_gather < r_gather_end)
		{
			// GLOBAL MEMORY access
			int32_t neighbor = CSR_C[r_gather];	
			int sum = 0; 
			int active = 0;

			// Status lookup 
			volatile int32_t visited = (predecessors[neighbor] != -1);

			// Adding in out_queue
			if(!visited)
			{
				predecessors[neighbor] = shared_vertex[warp_id]; 
				active = 1; 
			}	

			if(__any(active))
			{
				sum = warpScanSumDown(active); 
				int offset = sum-1;
				if(lane_id == 0)
					offset_warp[warp_id] = atomicAdd(n_out_queue,sum); 
				if(active && offset > -1)
					out_queue[offset_warp[warp_id]+offset] = neighbor; 
			}

			r_gather += 32; 
		} 
	}	
}


/* Global functions required by graph500.c */
/* declared extern "C" for CUDA to C compatibility */
__global__ void explore_CSR(	int32_t * CSR_R, int32_t * CSR_C, 
				BITMAP_TYPE * in_queue, int32_t * n_in_queue,
				BITMAP_TYPE * out_queue, int32_t * n_out_queue,
				int32_t * predecessors)
{
	int lane_id = threadIdx.x & 31; 
	int warp_id = threadIdx.x >> 5;
	int word = blockIdx.x*NWARPS+warp_id;
	BITMAP_TYPE val_in_queue = in_queue[word]; 

	if(val_in_queue == 0)
		return; 

	uint32_t range[3] = {0,0,0};
	int32_t id_vertex = -1;

	if(val_in_queue & 1 << lane_id)
	{
		id_vertex = word*BITMAP_WORD+lane_id;
		range[0] = CSR_R[id_vertex];
		range[1] = CSR_R[id_vertex+1];
		range[2] = range[1] - range[0]; 
	}
	//if(blockIdx.x == 20)
		//printf("val_in_queue(%u) thread(%d) lane_id(%d) warp_id(%d) id_vertex(%d) range(%d,%d,%d)\n",val_in_queue,threadIdx.x,lane_id,warp_id,id_vertex,range[0],range[1],range[2]);

	volatile __shared__ int comm[NWARPS][3];
	volatile __shared__ int shared_vertex[NWARPS];
	uint32_t sum; 

	while(__any(range[2]))
	{
		int32_t neighbor = -1; 
	
		if(range[2])
			comm[warp_id][0] = lane_id; 
	
		if(comm[warp_id][0] == lane_id)
		{
			comm[warp_id][1] = range[0];
			comm[warp_id][2] = range[1]; 
			range[2] = 0; 
			shared_vertex[warp_id] = id_vertex; 
		}

		//if(blockIdx.x == 20)
			//printf("thread(%d) lane_id(%d) warp_id(%d) id_vertex(%d) shared_vertex(%d)(%d,%d)\n",threadIdx.x,lane_id,warp_id,id_vertex,shared_vertex,comm[warp_id][0],comm[warp_id][1]);

		int r_gather = comm[warp_id][1] + lane_id; 
		int r_gather_end = comm[warp_id][2]; 
		while(r_gather < r_gather_end)	
		{
			sum = 0; 
			neighbor = CSR_C[r_gather];
 
			int32_t prev = atomicCAS(&(predecessors[neighbor]),-1,shared_vertex[warp_id]);
			if(prev == -1){
				uint32_t val_out_queue = 1 << neighbor%BITMAP_WORD;
				atomicOr(&(out_queue[neighbor/BITMAP_WORD]),val_out_queue);
				sum = 1;
			} 

			// Check visited 
			if(__any(sum))
			{
				sum = warpReduceSum(sum); 
				if(lane_id == 0)
					atomicAdd(n_out_queue,sum); 
			}		
			r_gather += 32; 
		}
	}
}

__global__ void setup_GPU(int32_t * predecessors, int64_t srcvtx, int32_t * in_queue)
{
	predecessors[srcvtx] = (int32_t)srcvtx;
	in_queue[0] = srcvtx;
}

/* Create the graph structure on the GPUs */
extern "C"
int create_graph_from_edgelist(struct packed_edge * IJ, int64_t nedge)
{
	//printf("create_graph_from_edgelist nedge(%" PRId64 ")\n",nedge);
	#pragma omp parallel 
	#pragma omp single 
	printf("%d threads\n", omp_get_num_threads());
	/* Each thread handle a GPU */		
	find_nv(IJ,nedge);

	/* Compute CSR representation */
	edgelist_to_CSR(IJ,nedge);

	return 0; 
}

extern "C"
int make_bfs_tree( int64_t *bfs_tree_out, int64_t *max_vtx_out, int64_t srcvtx)
{
	printf("\n");
	// TODO check this nv != maxvtx
	*max_vtx_out = maxvtx;
	h_n_in_queue = 1;
	HANDLE_ERROR(hipMemcpy(d_n_in_queue,&h_n_in_queue,sizeof(int32_t),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(d_predecessors,-1,sizeof(int32_t)*nv));
	setup_GPU<<<1,1>>>(d_predecessors,srcvtx,d_in_queue);
	//setup_GPU<<<(nv + (NWARPS/2))/NWARPS,NWARPS>>>(d_predecessors,srcvtx);
	
	int32_t iteration = 0;

	while(1)
	{
		if(iteration++ > 1 << 20)
		{
			fprintf(stderr,"Too many iterations(%d)\n",iteration);
			return -1;
		}

		dim3 dimGrid(h_n_in_queue/(NWARPS*32)+1,0,0); 		
		dim3 dimBlock(32*NWARPS ,0,0);
	
		printf("iteration(%2d) n_in_queue(%10d) nblocks(%4d) nthreads(%d) ",
			iteration,
			h_n_in_queue,
			dimGrid.x,
			dimBlock.x);
		fflush(stdout);
	
		HANDLE_ERROR(hipMemset(d_n_out_queue,0,sizeof(int32_t)));
		hipEventRecord(start);
		if(1)
		{
			printf(" CSR ");
			// Do not copy in_queue into out_queue, just swap the device pointers 
			if(iteration%2)
			{
				printf(" C1 ");
				expand_contract_merrill<<< dimGrid.x,dimBlock.x >>>(	
						d_CSR_R,d_CSR_C,d_in_queue,d_n_in_queue,
						d_out_queue,d_n_out_queue,d_predecessors);
			}else{
				printf(" C2 ");
				expand_contract_merrill<<< dimGrid.x,dimBlock.x >>>(	
						d_CSR_R,d_CSR_C,d_out_queue,d_n_in_queue,
						d_in_queue,d_n_out_queue,d_predecessors);
			}
		}/*else{
			printf(" CSC ");
			explore_CSC<<< (nwords + (NWARPS/2))/NWARPS, 32*NWARPS >>>(
						d_CSR_R,d_CSR_C,d_in_queue,d_n_in_queue,
						d_out_queue,d_n_out_queue,d_predecessors);
		}*/
		hipEventRecord(stop);
		HANDLE_ERROR(hipMemcpy(&h_n_out_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToHost));
		hipEventSynchronize(stop);
		float milliseconds = 0; 
		hipEventElapsedTime(&milliseconds,start,stop);
		printf("out_queue(%10d) time(%.4f)s \n",h_n_out_queue,milliseconds/1000);	
		if(h_n_out_queue == 0)
		{
			printf("BFS ended\n");
			break;
		}	
		/* Switch queues */
		//HANDLE_ERROR(hipMemcpy(d_in_queue,d_out_queue,sizeof(int32_t)*maxedg,hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(d_n_in_queue,d_n_out_queue,sizeof(int32_t),hipMemcpyDeviceToDevice));
		h_n_in_queue = h_n_out_queue;
	}

	HANDLE_ERROR(hipMemcpy(h_predecessors,d_predecessors,sizeof(int32_t)*nv,hipMemcpyDeviceToHost));
	#pragma omp parallel for
	for(unsigned int i = 0 ; i < nv ; ++i)
	{
		bfs_tree_out[i] = (int64_t)h_predecessors[i];		
		assert(bfs_tree_out[i] < nv);
		assert(bfs_tree_out[i] > -2);
	}
	return 0;
}

extern "C"
void destroy_graph()
{
	free(h_CSR_R);
	free(h_CSR_C);
	hipFree(d_CSR_R);
	hipFree(d_CSR_C);
	hipFree(d_in_queue);
	hipFree(d_out_queue);
}
